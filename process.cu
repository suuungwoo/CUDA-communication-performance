#include "hip/hip_runtime.h"
#include "process.cuh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

//カーネル　指定したスレッド数分だけ動く！
__global__ void kernel( int* pSrc1, int *pSrc2, int *pResult, int length )
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x; //自分のスレッドのindex

	if (idx >= length) {
		return; //配列をはみ出る場合は無視
	}

	pResult[idx] = pSrc1[idx] + pSrc2[idx]; //加算

	return;
}

void Process()
{
	int length = 1024; 
	size_t size = sizeof(int) * length; 
	

	//ホストメモリのポインタ
	int* pHostSrc1;		//数値１
	int* pHostSrc2;		//数値２
	int* pHostResult;	//加算結果
	//デバイスメモリのポインタ
	int* pDevSrc1;		//数値１
	int* pDevSrc2;		//数値２
	int* pDevResult;	//加算結果


	//ホストメモリの確保
	hipHostMalloc(&pHostSrc1, size);	
	hipHostMalloc(&pHostSrc2, size);	
	hipHostMalloc(&pHostResult, size);	
	//デバイスメモリの確保
	hipMalloc(&pDevSrc1, size);		
	hipMalloc(&pDevSrc2, size);		
	hipMalloc(&pDevResult, size);		

	//ホストメモリに適当な値を設定
	for (int n = 0; n < length; n++) {
		pHostSrc1[n] = n;
		pHostSrc2[n] = n;
	}

	//ホスト->デバイスへ数値を転送
	hipMemcpy(pDevSrc1, pHostSrc1, size, hipMemcpyHostToDevice);
	hipMemcpy(pDevSrc2, pHostSrc2, size, hipMemcpyHostToDevice);

	//カーネル実行
	dim3 block(128, 1, 1);
	dim3 grid((length + 128 - 1 ) / 128, 1, 1);　//length数だけスレッドが生成されるようにしている
	kernel <<<grid, block>>> (pDevSrc1, pDevSrc2, pDevResult, length);

	//デバイス->ホストへ結果を転送
	hipMemcpy(pHostResult, pDevResult, size, hipMemcpyDeviceToHost);

	//デバイスメモリの開放
	hipFree(pDevSrc1);
	hipFree(pDevSrc2);
	hipFree(pDevResult);
	//ホストメモリの開放
	hipHostFree(pHostSrc1);
	hipHostFree(pHostSrc2);
	hipHostFree(pHostResult);

	hipDeviceReset();
}
