#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda_rentime.h>

#define N 1024

__global__ void VecAdd(float *A, float *B, float *C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

int main() {
  float h_A[N], h_B[N], h_C[N];
  size_t size = N * sizeof(float);
  float *d_A;
  hipMalloc((void **)&d_A, size);
  float *d_B;
  hipMalloc((void **)&d_B, size);
  float *d_C;
  hipMalloc((void **)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  int threadsPerBlock = N;
  int threadsPerGrid = 1;

  VecAdd<<<threadsPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
